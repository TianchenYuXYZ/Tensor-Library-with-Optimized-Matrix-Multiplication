#include<stdio.h>
#include<string.h>
#include<pybind11/pybind11.h>
#include<pybind11/numpy.h>
#include<pybind11/stl.h>

namespace py = pybind11;

void touch_alloc(){
  int* tmp;
  hipMalloc(&tmp,1);
  hipFree(tmp);
}

PYBIND11_MODULE(touch_gpu,m){
  m.def("touch_alloc",&touch_alloc);
}

